#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "plugin.h"
#include "hip/hip_fp16.h"
#include "gatherNMSCustomOutputs.h"
#include <array>

// __half minus with fallback to float for old sm
inline __device__ __half minus_fb(const __half & a, const __half & b) {
#if __CUDA_ARCH__ >= 530
    return a - b;
#else
    return __float2half(__half2float(a) - __half2float(b));
#endif
}

// overload for float
inline __device__ float minus_fb(const float & a, const float & b) {
    return a - b;
}

template <typename T_BBOX>
__device__ T_BBOX saturate(T_BBOX v)
{
    return max(min(v, T_BBOX(1)), T_BBOX(0));
}

template <>
__device__ __half saturate(__half v)
{
#if __CUDA_ARCH__ >= 800
    return __hmax(__hmin(v, __half(1)), __half(0));
#elif __CUDA_ARCH__ >= 530
    return __hge(v, __half(1)) ? __half(1) : (__hle(v, __half(0)) ? __half(0) : v);
#else
    return max(min(v, float(1)), float(0));
#endif
}

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSCustomOutputs_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        const T_BBOX* landData,
        int* numDetections,
        T_BBOX* nmsedBoxes,
        T_BBOX* nmsedScores,
        T_BBOX* nmsedClasses,
        T_BBOX* nmsedLandmarks,
        bool clipBoxes,
        const T_SCORE scoreShift
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            nmsedClasses[i] = -1;
            nmsedScores[i] = 0;
            nmsedBoxes[i * 4] = 0;
            nmsedBoxes[i * 4 + 1] = 0;
            nmsedBoxes[i * 4 + 2] = 0;
            nmsedBoxes[i * 4 + 3] = 0;
            nmsedLandmarks[i * 10]     = 0;
            nmsedLandmarks[i * 10 + 1] = 0;
            nmsedLandmarks[i * 10 + 2] = 0;
            nmsedLandmarks[i * 10 + 3] = 0;
            nmsedLandmarks[i * 10 + 4] = 0;
            nmsedLandmarks[i * 10 + 5] = 0;
            nmsedLandmarks[i * 10 + 6] = 0;
            nmsedLandmarks[i * 10 + 7] = 0;
            nmsedLandmarks[i * 10 + 8] = 0;
            nmsedLandmarks[i * 10 + 9] = 0;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset) * 4;
            nmsedClasses[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedScores[i] = score;                                                        // confidence score
            nmsedScores[i] = minus_fb(nmsedScores[i], scoreShift);
            const int lankmarkId = (index % numPredsPerClass)*10;

            nmsedLandmarks[i * 10]     = (T_BBOX) landData[lankmarkId];
            nmsedLandmarks[i * 10 + 1] = (T_BBOX) landData[lankmarkId + 1];
            nmsedLandmarks[i * 10 + 2] = (T_BBOX) landData[lankmarkId + 2];
            nmsedLandmarks[i * 10 + 3] = (T_BBOX) landData[lankmarkId + 3];
            nmsedLandmarks[i * 10 + 4] = (T_BBOX) landData[lankmarkId + 4];
            nmsedLandmarks[i * 10 + 5] = (T_BBOX) landData[lankmarkId + 5];
            nmsedLandmarks[i * 10 + 6] = (T_BBOX) landData[lankmarkId + 6];
            nmsedLandmarks[i * 10 + 7] = (T_BBOX) landData[lankmarkId + 7];
            nmsedLandmarks[i * 10 + 8] = (T_BBOX) landData[lankmarkId + 8];
            nmsedLandmarks[i * 10 + 9] = (T_BBOX) landData[lankmarkId + 9];
            const T_BBOX xMin = bboxData[bboxId];
            const T_BBOX yMin = bboxData[bboxId + 1];
            const T_BBOX xMax = bboxData[bboxId + 2];
            const T_BBOX yMax = bboxData[bboxId + 3];
            // clipped bbox xmin
            nmsedBoxes[i * 4] = clipBoxes ? saturate(xMin) : xMin;
            // clipped bbox ymin
            nmsedBoxes[i * 4 + 1] = clipBoxes ? saturate(yMin) : yMin;
            // clipped bbox xmax
            nmsedBoxes[i * 4 + 2] = clipBoxes ? saturate(xMax) : xMax;
            // clipped bbox ymax
            nmsedBoxes[i * 4 + 3] = clipBoxes ? saturate(yMax) : yMax;
            atomicAdd(&numDetections[i / keepTopK], 1);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSCustomOutputs_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    const void* landData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    void* nmsedLandmarks,
    bool clipBoxes,
    const float scoreShift
    )
{
    hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSCustomOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData, (T_BBOX*) landData,
                                                                           (int*) numDetections,
                                                                           (T_BBOX*) nmsedBoxes,
                                                                           (T_BBOX*) nmsedScores,
                                                                           (T_BBOX*) nmsedClasses,
                                                                           (T_BBOX*) nmsedLandmarks,
                                                                           clipBoxes,
                                                                           T_SCORE(scoreShift)
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSCustomOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               const void*,
                               void*,
                               void*,
                               void*,
                               void*,
                               void*,
                               bool,
                               const float);
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::array<nmsOutLaunchConfig, 2> nmsOutLCOptions = {
  nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSCustomOutputs_gpu<float, float>),
  nmsOutLaunchConfig(DataType::kHALF, DataType::kHALF, gatherNMSCustomOutputs_gpu<__half, __half>)
};

pluginStatus_t gatherNMSCustomOutputs(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DataType DT_BBOX,
    const DataType DT_SCORE,
    const void* indices,
    const void* scores,
    const void* bboxData,
    const void* landData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    void* nmsedLandmarks,
    bool clipBoxes,
    const float scoreShift
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutLCOptions.size(); ++i)
    {
        if (lc == nmsOutLCOptions[i])
        {
            DEBUG_PRINTF("gatherNMSCustomOutputs kernel %d\n", i);
            return nmsOutLCOptions[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          landData,
                                          numDetections,
                                          nmsedBoxes,
                                          nmsedScores,
                                          nmsedClasses,
                                          nmsedLandmarks,
                                          clipBoxes,
                                          scoreShift
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}
